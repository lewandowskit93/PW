#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <iostream>

__device__ int getVertexEdgesCountDevice(int *edges_matrix,const int n,bool symetric, int x)
{
  int edges_count = 0;
  for (int i = (symetric ? x + 1 : 0); i < n; ++i)
  {
    if (x == i)continue;
    if (edges_matrix[x*n+i])++edges_count;
  }
  return edges_count;
}

__global__ void countEdgesKernel(int *edges_matrix, bool symetric,int* edges_count,const int n)
{
    int sub_count = 0;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int c_units = gridDim.x*blockDim.x;
    for (int i = id; i < n; i += c_units)
    {
      sub_count += getVertexEdgesCountDevice(edges_matrix, n, symetric, i);
    }
    atomicAdd(edges_count, sub_count);
}

int countEdgesCuda(int blocks, int threads, int *edges_matrix, bool symetric, const int n)
{
  int *dev_edges_matrix = 0;
  int *dev_edges_count = 0;
  hipSetDevice(0);

  hipMalloc((void**)&dev_edges_count, sizeof(int));
  int edges_count = 0;
  hipMemcpy(dev_edges_count, &edges_count, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**)&dev_edges_matrix, sizeof(int)*n*n);
  hipMemcpy(dev_edges_matrix, edges_matrix, sizeof(int)*n*n, hipMemcpyHostToDevice);

  countEdgesKernel << <blocks, threads >> >(dev_edges_matrix, true, dev_edges_count, n);

  hipMemcpy(&edges_count, dev_edges_count, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_edges_count);

  hipFree(dev_edges_matrix);

  return edges_count;
}

int main()
{
  int blocks;
  int threads;
    int n;
    bool symetric;
    int nr_of_edges;
    std::cin >> blocks;
    std::cin >> threads;
    std::cin >> n;
    std::cin >> symetric;
    std::cin >> nr_of_edges;
    int *edges_matrix = new int[n*n];
    for (int i = 0; i < n; ++i)
    {
      for (int j = 0; j < n; ++j)
      {
        edges_matrix[i*n+j] = 0;
      }
    }
    for (int i = 0; i < nr_of_edges; ++i)
    {
      int x, y;
      std::cin >> x >> y;
      edges_matrix[x*n+y] = 1;
      if (symetric)edges_matrix[y*n+x] = 1;
    }

    std::cout << countEdgesCuda(blocks,threads,edges_matrix, symetric, n);

    return 0;
}
